#include "hip/hip_runtime.h"
#include <windows.h>
#include "../DeviceBuffer.h"
#include "../Fill.h"
#include "../Nutty.h"
#include <hip/hip_runtime.h>

//__global__ 

int main(void)
{
#if 1
    _CrtSetDbgFlag (_CRTDBG_ALLOC_MEM_DF | _CRTDBG_LEAK_CHECK_DF);
    _CrtSetReportMode(_CRT_ERROR, _CRTDBG_MODE_FILE);
    _CrtSetReportFile(_CRT_ERROR, _CRTDBG_FILE_STDERR);
#endif

    //create nutty
    nutty::Init();

    size_t freeMemory, totalMemory;
    hipMemGetInfo(&freeMemory, &totalMemory);

    printf("%u %u\n", totalMemory, freeMemory);
    
    //hipMalloc(

    nutty::DeviceBuffer<char> memory0(1024 * 1024 * 2);
    nutty::DeviceBuffer<char> memory1(1024 * 1024 * 1024);
    nutty::DeviceBuffer<char> memory2(1024 * 1024 * 1024);
    //nutty::DeviceBuffer<int> memory3(1024 * 1024 * 500);

    nutty::Fill(memory0.Begin(), memory0.End(), (char)0);
    nutty::Fill(memory1.Begin(), memory1.End(), (char)0);
    //nutty::Fill(memory2.Begin(), memory2.End(), 0);
    //nutty::Fill(memory3.Begin(), memory3.End(), 0);
    //release nutty
    nutty::Release();

    return 0;
}