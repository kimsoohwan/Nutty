#include <windows.h>
#include "../Nutty.h"
#include "../Fill.h"
#include "../Sort.h"
#include <sstream>
#include <fstream>
#include "../Inc.h"
#include "../ForEach.h"
#include "../Functions.h"
#include "../cuTimer.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>

uint c = 0;

void print(const int& t)
{
    std::stringstream ss;
    ss << t;
    ss << " "; 
    OutputDebugStringA(ss.str().c_str());
   // if((++c % 16) == 0) OutputDebugStringA("\n");
}

struct sortinfo
{
    uint pos;
    int a;
    int b;
};

template < typename IT>
bool checkSort(IT& b, uint size, sortinfo* si)
{
    nutty::HostBuffer<int> cpy(b.Size());
    nutty::Copy(cpy.Begin(), b.Begin(), size);
    auto it = cpy.Begin();
    int i = 0;
    int _cc = 0;
    while(it != cpy.End())
    {
        int cc = *it;
        if(cc < i)
        {
            //si->pos = _cc;
            //si->a = i;
            //si->b = cc;
            /*std::stringstream m;
            m << cc;
            m << " ";
            m << i << ", ";
            m << _cc;
            OutputDebugStringA(m.str().c_str());
            OutputDebugStringA("\n");*/
            return false;
        }
        i = cc;
        it++;
        _cc++;
    }
    return true;
}

int main(void)
{
    #define _CRTDBG_MAP_ALLOC

#if 1
    _CrtSetDbgFlag (_CRTDBG_ALLOC_MEM_DF | _CRTDBG_LEAK_CHECK_DF);
    _CrtSetReportMode(_CRT_ERROR, _CRTDBG_MODE_FILE);
    _CrtSetReportFile(_CRT_ERROR, _CRTDBG_FILE_STDERR);
#endif

    nutty::Init();
    /*
    dim3 grid = 1 << 32 - 1;
    dim3 block = 512;
    nutty::DeviceBuffer<uint> ptr;
    ptr.Resize(grid.x * block.x);
    testKernel<<<grid, block >>>(ptr.Begin()());*/

    DEVICE_SYNC_CHECK();

    hipDeviceProp_t props;

    hipGetDeviceProperties(&props, 0);
    
    std::ofstream profile("sorting_profile.txt");

    profile << "maxgridx=" << props.maxGridSize[0] << "\n";

    profile << "Elems\nTime\n\n";

    std::vector<double> times;

    nutty::cuTimer timer;

    int runs = 1;
    uint startBit = 8;
    uint maxBit = 25;
    uint ivalStep = 1e6;
    srand(0);

    profile << "runs: " << runs << "\n\n";

    for(uint i = startBit; i < maxBit; ++i)
    {
        times.push_back(0);
        profile << (1 << i) << "\n";
    }

    bool error = false;
    for(int k = 0; k < runs; k++)
    {
        if(error)
        {
            break;
        }

        for(int i = startBit; i < maxBit; ++i)
        {
            uint elems = 1 << i;

//             nutty::DeviceBuffer<uint> a(elems);
// 
//             nutty::Fill(a.Begin(), a.End(), rand);

            timer.Start();

            //run_qsort(a.Begin()(), elems);
            //nutty::Sort(a.Begin(), a.End(), nutty::BinaryDescending<int>());

//             thrust::device_vector<int> ta(elems);
//             thrust::generate(ta.begin(), ta.end(), rand);
//             thrust::sort(ta.begin(), ta.end());

            hipError_t err = hipDeviceSynchronize();

            timer.Stop();

            if(err != hipSuccess)
            {
                profile << elems << " -> " << hipGetErrorString(err) << "\n\n";
                error = true;
                break;
            }
 
            times[i - startBit] += timer.GetMillis() / 1000.0; 
        }
    }

    for(int i = 0; i < maxBit - startBit; ++i)
    {
        times[i] = times[i] / (double) runs;
    }

    profile << "\n\n";

    for(auto& it : times)
    {
        profile << it << "\n";
    }

    profile.close();

    nutty::Release();

    return 0;
}